#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <vector>


#define BLO 512
#define BB 250000
#define uint unsigned int

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)

__global__ void digit_of_a_number(uint* dev_data, int size, int i, uint* dev_b) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int offsetx = blockDim.x * gridDim.x;
	for (int j = idx; j < size; j += offsetx) {
		dev_b[j] = (dev_data[j] >> i) & 1;
	}
}

#define _index(i) ((i) + ((i) >> 8))

__global__ void par_scan(uint* dev_b, int size, uint* displace) {
	__shared__ int sdata[BLO + 120];
	int idx = threadIdx.x;
	int offsetx = (gridDim.x * blockIdx.y + blockIdx.x) * BLO;
	uint tmp;
	int index, of, s;
	if (offsetx + idx < size) {
		sdata[_index(idx)] = dev_b[offsetx + idx];
		for (s = 1; s <= BLO / 2; s <<= 1) {
			__syncthreads();
			of = s - 1;
			index = 2 * s * idx;
			if (index < BLO)
				sdata[_index(of + index + s)] += sdata[_index(of + index)];
		}
		if (idx == 0) {
			displace[gridDim.x * blockIdx.y + blockIdx.x] = sdata[_index(BLO - 1)];
			sdata[_index(BLO - 1)] = 0;
		}
		for (s = BLO / 2; s > 0; s >>= 1) {
			__syncthreads();
			of = s - 1;
			index = 2 * s * idx;
			if (index < BLO) {
				tmp = sdata[_index(of + index + s)];
				sdata[_index(of + index + s)] += sdata[_index(of + index)];
				sdata[_index(of + index)] = tmp;
			}
		}
		__syncthreads();
		dev_b[offsetx + idx] = sdata[_index(idx)];
	}
}

__global__ void back(uint* dev_b, int size, int blocks, uint* displace) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = gridDim.x * blockDim.x;
	for (int i = idx; i < size; i += offset) {
		dev_b[i] += displace[i / BLO];
	}
}


void scan(uint* dev_b, int size) {
	int blocks = (size - 1) / BLO + 1;
	uint* displace;
	hipMalloc(&displace, blocks * sizeof(uint));
	if (blocks < BLO) {
		par_scan << <dim3(BLO, BLO), dim3(BLO) >> > (dev_b, size, displace);
		std::cerr << blocks << "(1) ";
		CSC(hipGetLastError());
	}
	else {
		par_scan << <dim3(BLO, BLO), dim3(BLO) >> > (dev_b, size, displace);
		std::cerr << blocks << "(2) ";
		CSC(hipGetLastError());
	}
	if (blocks == 1) {
		CSC(hipFree(displace));
		return;
	}
	scan(displace, blocks);
	back << <BLO, BLO >> > (dev_b, size, blocks, displace);
	std::cerr << blocks << "(3) ";
	CSC(hipGetLastError());
	CSC(hipFree(displace));
}

__global__ void radix_sort(uint* dev_s, int size, int i, uint* dev_data, uint* dev_b) {
	uint ter = 0;
	if (((dev_s[size - 1] >> i) & 1) == 1) {
		ter = 1;
	}
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int offsetx = blockDim.x * gridDim.x;
	for (int j = idx; j < size; j += offsetx) {
		if (((dev_s[j] >> i) & 1) == 0) {
			dev_data[j - dev_b[j]] = dev_s[j];
		}
		else {
			dev_data[dev_b[j] + (size - (dev_b[size - 1] + ter))] = dev_s[j];
		}
	}
}

int main() {
	std::ios_base::sync_with_stdio(0);
	std::cin.tie(0);
	int size;
	fread(&size, sizeof(int), 1, stdin);
	 // std::cin >> size;
	if (size == 0) {
		return 0;
	}
	uint* data = new uint[size];
	fread(data, sizeof(uint), size, stdin);
	 // for (int i = 0; i < size; ++i) {
	 // 	std::cin >> data[i];
	 // }
	std::cerr << size << " ";
	//for (int i = 0; i < size; ++i) {
	//	std::cerr << data[i] << " ";
	//}
	uint* dev_data, * dev_s, * dev_b, * tmp;
	CSC(hipMalloc(&dev_data, size * sizeof(uint)));
	CSC(hipMalloc(&dev_s, size * sizeof(uint)));
	CSC(hipMalloc(&dev_b, size * sizeof(uint)));
	CSC(hipMemcpy(dev_data, data, size * sizeof(uint), hipMemcpyHostToDevice));
	CSC(hipMemcpy(dev_s, data, size * sizeof(uint), hipMemcpyHostToDevice));
	for (int j = 0; j < 32; j++) {

		std::cerr << j << " " << "!!!!!!!!" << std::endl;

		digit_of_a_number << <BLO, BLO >> > (dev_data, size, j, dev_b);
		CSC(hipGetLastError());
		//	CSC(hipMemcpy(data, dev_b, size * sizeof(uint), hipMemcpyDeviceToHost));
		//	for (int i = 0; i < size; ++i) {
		//		std::cout << data[i] << " ";
		//	}
		//	std::cout << std::endl;
		scan(dev_b, size);

		//	CSC(hipMemcpy(data, dev_b, size * sizeof(uint), hipMemcpyDeviceToHost));
		//	for (int i = 0; i < size; ++i) {
		//		std::cout << data[i] << " ";
		//	}
		//	std::cout << std::endl;
		//	CSC(hipMemcpy(data, dev_data, size * sizeof(uint), hipMemcpyDeviceToHost));
		//	for (int i = 0; i < size; ++i) {
		//		std::cout << data[i] << " ";
		//	}
		//	std::cout << std::endl;


		tmp = dev_data;
		dev_data = dev_s;
		dev_s = tmp;


		radix_sort << <BLO, BLO >> > (dev_s, size, j, dev_data, dev_b);


		//	CSC(hipMemcpy(data, dev_data, size * sizeof(uint), hipMemcpyDeviceToHost));
		//	for (int i = 0; i < size; ++i) {
		//		std::cout << data[i] << " ";
		//	}
		//	std::cout << std::endl;
		//	std::cout << std::endl;
		//	std::cout << std::endl;
		CSC(hipGetLastError());
	}

	//scan(dev_data, size);


	CSC(hipMemcpy(data, dev_data, size * sizeof(uint), hipMemcpyDeviceToHost));
	fwrite(data, sizeof(uint), size, stdout);
	 // for (int i = 0; i < size; ++i) {
	 // 	std::cout << data[i] << " ";
	 // }
	hipFree(dev_data);
	hipFree(dev_b);
	hipFree(dev_s);
	free(data);
	return 0;
}

// 11
// 0 1 1 0 1 0 0 1 1 0 1
