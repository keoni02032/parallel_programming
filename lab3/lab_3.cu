
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <vector>

typedef struct {
	int x;
	int y;
} point;

typedef struct {
	float x = 0;
	float y = 0;
	float z = 0;
} cord_float;

__constant__ float avg[32][3];
__constant__ float norm_avg[32][3];

__global__ void kernel(uchar4* pixels, int w, int h, int count_clases) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
	int x, y, i, max_p;
	cord_float rgb;
	float max_val;
	float result[32];
	for(y = idy; y < h; y += offsety) {
		for(x = idx; x < w; x += offsetx) {
			uchar4 p;
			p.x = pixels[y * w + x].x;
			p.y = pixels[y * w + x].y;
			p.z = pixels[y * w + x].z;
			for (i = 0; i < count_clases; ++i) {
				rgb.x = p.x * norm_avg[i][0];
				rgb.y = p.y * norm_avg[i][1];
				rgb.z = p.z * norm_avg[i][2];
				result[i] = rgb.x + rgb.y + rgb.z;
			}
			max_val = result[0];
			max_p = 0;
			for (i = 1; i < count_clases; ++i) {
				if (max_val < result[i]) {
					max_val = result[i];
					max_p = i;
				}
			}
			pixels[y * w + x].w = (char)max_p;
		}
	}
}

int main(int argc, const char* argv[])
{
	std::string in, out;
	int w, h, count_clases, size;
	std::cin >> in >> out >> count_clases;
	std::vector<std::vector<point>> vv(count_clases);
	for (int i = 0; i < count_clases; ++i) {
		std::cin >> size;
		vv[i].resize(size);
		for (int j = 0; j < size; ++j) {
			point t;
			std::cin >> t.x >> t.y;
			vv[i][j] = t;
		}
	}
	FILE *fp = fopen(in.c_str(), "rb");
	fread(&w, sizeof(int), 1, fp);
	fread(&h, sizeof(int), 1, fp);
	uchar4 *data;
	data = (uchar4*)malloc(sizeof(uchar4) * w * h);
	fread(data, sizeof(uchar4), w * h, fp);
	fclose(fp);uchar4 *dev_out;
	hipMalloc(&dev_out, sizeof(uchar4) * w * h);
	hipMemcpy(dev_out, data, sizeof(uchar4) * w * h, hipMemcpyHostToDevice);
	

	float cpu_avg[32][3];
	for (int i = 0; i < count_clases; ++i) {
		point c;
		cpu_avg[i][0] = 0;
		cpu_avg[i][1] = 0;
		cpu_avg[i][2] = 0;
		for (int j = 0; j < vv[i].size(); ++j) {
			c.x = vv[i][j].x;
			c.y = vv[i][j].y;
			cpu_avg[i][0] += data[c.y * w + c.x].x;
			cpu_avg[i][1] += data[c.y * w + c.x].y;
			cpu_avg[i][2] += data[c.y * w + c.x].z;
		}
		cpu_avg[i][0] /= vv[i].size();
		cpu_avg[i][1] /= vv[i].size();
		cpu_avg[i][2] /= vv[i].size();
	}
	
	float norm_cpu_avg[32][3];
	for (int i = 0; i < count_clases; ++i) {
		norm_cpu_avg[i][0] = cpu_avg[i][0] / std::sqrt(cpu_avg[i][0] * cpu_avg[i][0] + cpu_avg[i][1] * cpu_avg[i][1] + cpu_avg[i][2] * cpu_avg[i][2]);
		norm_cpu_avg[i][1] = cpu_avg[i][1] / std::sqrt(cpu_avg[i][0] * cpu_avg[i][0] + cpu_avg[i][1] * cpu_avg[i][1] + cpu_avg[i][2] * cpu_avg[i][2]);
		norm_cpu_avg[i][2] = cpu_avg[i][2] / std::sqrt(cpu_avg[i][0] * cpu_avg[i][0] + cpu_avg[i][1] * cpu_avg[i][1] + cpu_avg[i][2] * cpu_avg[i][2]);
	}
	
	hipMemcpyToSymbol(HIP_SYMBOL(avg), cpu_avg, sizeof(float) * 32 * 3);
	hipMemcpyToSymbol(HIP_SYMBOL(norm_avg), norm_cpu_avg, sizeof(float) * 32 * 3);
    kernel<<<dim3(32, 32), dim3(32, 32)>>>(dev_out, w, h, count_clases);
	hipMemcpy(data, dev_out, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost);
	hipFree(dev_out);
	FILE *fp1;
	fp1 = fopen(out.c_str(), "wb");
	fwrite(&w, sizeof(int), 1, fp1);
	fwrite(&h, sizeof(int), 1, fp1);
	fwrite(data, sizeof(uchar4), w * h, fp1);
	fclose(fp1);
	free(data);
	return 0;
}